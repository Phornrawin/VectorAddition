#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__global__ void vectorAdd(int *d_a, int *d_b, int *d_c, int n) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int b = blockIdx.x ;

	if (i >= n) {
		return;
	}
	d_c[i] = d_a[i] + d_b[i];

	/*for (int i = 0; i < n; i++) {
		d_c[i] = d_a[i] + d_b[i];
		printf("C[%d] = %d from thread = %d\n", i, d_c[i], threadIdx.x);
	}*/

	printf("C[%d] = %d thread = %d block = %d \n", i, d_c[i], i, b);
}
int main() {
	const int N = 3000;
	
	int h_a[N];
	int h_b[N];

	int h_c[N];

	for (int i = 0; i < N; i++) {
		h_a[i] = i;
	}

	for (int i = 0; i < N; i++) {
		h_b[i] = N-i;
	}

	//Part1
	int *d_a, *d_b, *d_c;
	hipMalloc((void**) &d_a, N * sizeof(int));
	hipMalloc((void**) &d_b, N * sizeof(int));
	hipMalloc((void**) &d_c, N * sizeof(int));

	hipMemcpy(d_a, &h_a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &h_b, N * sizeof(int), hipMemcpyHostToDevice);

	//define DUDA Timer
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//start Cuda timer
	hipEventRecord(start, 0);

	//Part2
	int blockNumber = ceil(N / 1024);
	vectorAdd<<<blockNumber,1024>>>(d_a, d_b, d_c, N);
	
	//stop Cuda timer
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	//compute elapsed time
	float time;
	hipEventElapsedTime(&time, start, stop);

	//Part3
	hipMemcpy(&h_c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	//report time in kernel
	printf("Time in kernel = %f ms \n", time);

	return 0;
}